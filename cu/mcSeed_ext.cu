
#include <hip/hip_runtime.h>
#define MAG(a,b) (sqrt((a)*(a)+(b)*(b)))
#define TWO (2147483648.0)

/* 
 * linear congruent generator for device side random number calculations
*/ 
__device__ int lcg(int seed)
{
    return seed*1103515245+12345 % 2147483648;
}

/* seeds individual spins for monte carlo simulation into onlyl extra-axonal 
 * space
 *
 *   xi - output x,y locations
 *   rn1 - seeds for computation of x,y
 *   rn2 - seeds for determining M0r
 *   roi - outer,inner radii
 *   xo - x0,y0 for axons
 *   nA - number of axons
 *   Lx - boundary of geometry
 *   M0r - ratio of myelin to non-myelin density
 *   N - number of spins
 */
__global__ void mcSeed_ext(float *xi, float *rn1, float *rn2, float* roi, 
        float* xo, int nA, float Lx, float M0r, int N) 
{
	int p = threadIdx.x + blockDim.x*blockIdx.x;
    
    if (p<N)
    {
        float r1, r2, r3, xt, yt;
        int m;
        
        r1 = rn1[p];
        r2 = rn1[p+N];
        r3 = rn2[p];
        
        xt = r1*Lx;
        yt = r2*Lx;
        
        int state=1;

        // loop until the spin is placed
        while (state)
        {
            state = 0;
            
            // Is this spin within an axon?
            for (m=0;m<nA;m++) 
            {
                if ( MAG(xt-xo[m],yt-xo[m+nA])<roi[m] )     
                {
                    // in side axon or myelin, reseeding...
                    state = 1;
                    r1 = abs(lcg(round(r1*TWO)))/TWO;
                    r2 = abs(lcg(round(r2*TWO)))/TWO;
                    r3 = abs(lcg(round(r3*TWO)))/TWO;
                    
                    xt = r1*Lx;
                    yt = r2*Lx;
                }
            }
        }
    
        xi[p] = xt;
        xi[p+N] = yt;
    }
}
